#include "hip/hip_runtime.h"
#include <math.h>
#include <ATen/ATen.h>
#include <c10/cuda/CUDAGuard.h>
#include <THC/THCAtomics.cuh>
#include "utils.h"

#define PRIVATE_CASE_TYPE_AND_VAL(ENUM_TYPE, TYPE, TYPE_NAME, VAL, ...) \
  case ENUM_TYPE: { \
    using TYPE_NAME = TYPE; \
    const int num_threads = VAL; \
    return __VA_ARGS__(); \
  }


#define DISPATCH_INPUT_TYPES(TYPE, TYPE_NAME, SCOPE_NAME, ...) \
  [&] { \
    switch(TYPE) \
    { \
      PRIVATE_CASE_TYPE_AND_VAL(at::ScalarType::Float, float, TYPE_NAME, 1024, __VA_ARGS__) \
      PRIVATE_CASE_TYPE_AND_VAL(at::ScalarType::Double, double, TYPE_NAME, 512, __VA_ARGS__) \
      default: \
        AT_ERROR(#SCOPE_NAME, " not implemented for '", toString(TYPE), "'"); \
    } \
  }()


namespace primitive {

struct my_float3 {
    float x, y, z;
    __device__ __forceinline__ float operator[](int i) const { 
        switch (i) {
            case 0: return x;
            case 1: return y;
            case 2: return z;
        };
        return x;
    };
    __device__ __forceinline__ float& operator[](int i) {
        switch (i) {
            case 0: return x;
            case 1: return y;
            case 2: return z;
        };
        return x;
    }
};

struct my_double3 {
    double x, y, z;
    __device__ __forceinline__ double operator[](int i) const { 
        switch (i) {
            case 0: return x;
            case 1: return y;
            case 2: return z;
        };
        return x;
    };
    __device__ __forceinline__ double& operator[](int i) {
        switch (i) {
            case 0: return x;
            case 1: return y;
            case 2: return z;
        };
        return x;
    }
};

__device__ __forceinline__ my_float3 make_my_float3(float x, float y, float z)
{
  my_float3 t; t.x = x; t.y = y; t.z = z; return t;
}

__device__ __forceinline__ my_double3 make_my_double3(double x, double y, double z)
{
  my_double3 t; t.x = x; t.y = y; t.z = z; return t;
}

template<typename T>
struct ScalarTypeToVec3 { using type = void; };
template <> struct ScalarTypeToVec3<float> { using type = my_float3; };
template <> struct ScalarTypeToVec3<double> { using type = my_double3; };

template<typename T>
struct Vec3TypeToScalar { using type = void; };
template <> struct Vec3TypeToScalar<my_float3> { using type = float; };
template <> struct Vec3TypeToScalar<my_double3> { using type = double; };


__device__ __forceinline__ my_float3 make_vector(float x, float y, float z) {
  return make_my_float3(x, y, z);
}

__device__ __forceinline__ my_double3 make_vector(double x, double y, double z) {
  return make_my_double3(x, y, z);
}

template <typename vector_t>
__device__ __forceinline__ typename Vec3TypeToScalar<vector_t>::type dot(vector_t a, vector_t b) {
  return a.x * b.x + a.y * b.y + a.z * b.z ;
}

template<typename scalar_t, typename vector_t>
__device__ __forceinline__ scalar_t dot2(vector_t v) {
  return dot<scalar_t, vector_t>(v, v);
}

template<typename scalar_t>
__device__ __forceinline__ scalar_t clamp(scalar_t x, scalar_t a, scalar_t b) {
  return max(a, min(b, x));
}

template<typename vector_t>
__device__ __forceinline__ vector_t clamp_vec(vector_t x, vector_t a, vector_t b) {
  return make_vector(clamp(x.x, a.x, b.x), clamp(x.y, a.y, b.y), clamp(x.z, a.z, b.z));
}

template<typename scalar_t>
__device__ __forceinline__ int sign(scalar_t a) {
  if (a <= 0) {return -1;}
  else {return 1;}
}

template<typename scalar_t, typename vector_t>
__device__ __forceinline__ vector_t operator* (vector_t a, scalar_t b) {
  return make_vector(a.x * b, a.y * b, a.z * b);
}

template<typename vector_t>
__device__ __forceinline__ vector_t operator* (vector_t a, vector_t b) {
  return make_vector(a.x * b.x, a.y * b.y, a.z * b.z);
}

template<typename scalar_t, typename vector_t>
__device__ __forceinline__ vector_t operator+ (vector_t a, scalar_t b) {
  return make_vector(a.x + b, a.y + b, a.z + b);
}

template<typename vector_t>
__device__ __forceinline__ vector_t operator+ (vector_t a, vector_t b) {
  return make_vector(a.x + b.x, a.y + b.y, a.z + b.z);
}

template<typename scalar_t, typename vector_t>
__device__ __forceinline__ vector_t operator- (vector_t a, scalar_t b) {
  return make_vector(a.x - b, a.y - b, a.z - b);
}

template<typename vector_t>
__device__ __forceinline__ vector_t operator- (vector_t a, vector_t b) {
  return make_vector(a.x - b.x, a.y - b.y, a.z - b.z);
}

template<typename vector_t>
__device__ __forceinline__ vector_t operator- (vector_t a) {
  return make_vector(-a.x, -a.y, -a.z);
}

template<typename scalar_t, typename vector_t>
__device__ __forceinline__ vector_t operator/ (vector_t a, scalar_t b) {
  return make_vector(a.x / b, a.y / b, a.z / b);
}

template<typename vector_t>
__device__ __forceinline__ vector_t operator/ (vector_t a, vector_t b) {
  return make_vector(a.x / b.x, a.y / b.y, a.z / b.z);
}

template<typename vector_t>
__device__ __forceinline__ vector_t abs_vec(vector_t a) {
    return make_vector(abs(a.x), abs(a.y), abs(a.z));
}

template<typename scalar_t, typename vector_t>
__device__ __forceinline__ vector_t max_vec(vector_t a, scalar_t b) {
    return make_vector(max(a.x, b), max(a.y, b), max(a.z, b));
}

template<typename scalar_t, typename vector_t>
__device__ __forceinline__ vector_t min_vec(vector_t a, scalar_t b) {
    return make_vector(min(a.x, b), min(a.y, b), min(a.z, b));
}

template<typename scalar_t, typename vector_t>
__device__ __forceinline__ scalar_t min_vec(vector_t a) {
    return min(a.x, min(a.y, a.z));
}

template<typename scalar_t, typename vector_t>
__device__ __forceinline__ scalar_t max_vec(vector_t a) {
    return max(a.x, max(a.y, a.z));
}

template<typename scalar_t>
__device__ __forceinline__ scalar_t square(scalar_t a) {
    return a * a;
}


template<typename scalar_t, typename vector_t>
__global__ void box_distance_forward_cuda_kernel(
    const vector_t* points,
    const vector_t* box,
    int num_points,
    scalar_t* distances,
    bool* dis_signs,
    vector_t* closest_points) {
    vector_t vbox = *box;
    for (int point_id = threadIdx.x + blockIdx.x * blockDim.x; point_id < num_points; point_id += blockDim.x * gridDim.x) {
        vector_t q = abs_vec(points[point_id]) - vbox;
        vector_t q_clamped = max_vec(q, scalar_t(0));
        distances[point_id] = dot(q_clamped, q_clamped) + square(min(max_vec<scalar_t, vector_t>(q), scalar_t(0)));
        dis_signs[point_id] = (q.x > 0) || (q.y > 0) || (q.z > 0);
        closest_points[point_id] = clamp_vec(points[point_id], - vbox, vbox);
        if (!dis_signs[point_id]) {
            int closest_face = 0;
            if (q[1] > q[closest_face]) closest_face = 1;
            if (q[2] > q[closest_face]) closest_face = 2;
            closest_points[point_id][closest_face] = vbox[closest_face] * sign(points[point_id][closest_face]);
        }
    }
}


template<typename scalar_t, typename vector_t>
__global__ void box_distance_backward_cuda_kernel(
    const scalar_t* grad_dist,
    const vector_t* points,
    const vector_t* clst_points,
    int num_points,
    vector_t* grad_points) {
    for (int point_id = threadIdx.x + blockIdx.x * blockDim.x; point_id < num_points; point_id += blockDim.x * gridDim.x) {
        // scalar_t grad_out = 2. * grad_dist[point_id];
        // vector_t dist_vec = points[point_id] - clst_points[point_id];
        // dist_vec = dist_vec * grad_out;
        // grad_points[point_id] = dist_vec;
        grad_points[point_id] = (points[point_id] - clst_points[point_id]) * (scalar_t(2) * grad_dist[point_id]);
    }
}

void box_distance_forward_cuda_impl(
    at::Tensor points, 
    at::Tensor box, 
    at::Tensor distances, 
    at::Tensor dis_signs, 
    at::Tensor closest_points) {
    const int num_threads = 512;
    const int num_points = points.size(0);
    const int num_blocks = (num_points + num_threads - 1) / num_threads;
    using scalar_t = float;
    using vector_t = ScalarTypeToVec3<scalar_t>::type;
    const at::cuda::OptionalCUDAGuard device_guard(at::device_of(points));
    box_distance_forward_cuda_kernel<scalar_t, vector_t><<<num_blocks, num_threads>>>(
        reinterpret_cast<vector_t*>(points.data_ptr<scalar_t>()),
        reinterpret_cast<vector_t*>(box.data_ptr<scalar_t>()),
        points.size(0),
        distances.data_ptr<scalar_t>(),
        dis_signs.data_ptr<bool>(),
        reinterpret_cast<vector_t*>(closest_points.data_ptr<scalar_t>()));
    CUDA_CHECK(hipGetLastError());
}

void box_distance_backward_cuda_impl(
    at::Tensor grad_distances, 
    at::Tensor points, 
    at::Tensor closest_points, 
    at::Tensor grad_points) {

    const int num_threads = 512;
    const int num_points = points.size(0);
    const int num_blocks = (num_points + num_threads - 1) / num_threads;
    using scalar_t = float;
    using vector_t = ScalarTypeToVec3<scalar_t>::type;
    const at::cuda::OptionalCUDAGuard device_guard(at::device_of(points));
    box_distance_backward_cuda_kernel<scalar_t, vector_t><<<num_blocks, num_threads>>>(
        grad_distances.data_ptr<scalar_t>(),
        reinterpret_cast<vector_t*>(points.data_ptr<scalar_t>()),
        reinterpret_cast<vector_t*>(closest_points.data_ptr<scalar_t>()),
        points.size(0),
        reinterpret_cast<vector_t*>(grad_points.data_ptr<scalar_t>()));
    CUDA_CHECK(hipGetLastError());
}

}  // namespace primitive

#undef PRIVATE_CASE_TYPE_AND_VAL
#undef DISPATCH_INPUT_TYPES
